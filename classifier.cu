#include "hip/hip_runtime.h"
#include <math.h>
#include <mpi.h>
#include <cassert>

#include "classifier.h"
#include "util.h"

// static int mpi_rank;

#define CHECK_CUDA(call)                                              \
  do {                                                                \
    hipError_t status_ = call;                                       \
    if (status_ != hipSuccess) {                                     \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));                           \
      exit(EXIT_FAILURE);                                             \
    }                                                                 \
  } while (0)

#define CEIL_DIV(x, y) (((x) + (y)-1) / (y))
#define BATCH 4
// Multi-dimensional matrix containing fp32 elements
struct Tensor {
  Tensor(std::vector<int> shape_);
  Tensor(std::vector<int> shape_, float *buf_);
  ~Tensor();
  int num_elem();
  void fill_zeros();

  float *buf = nullptr;
  float *gbuf = nullptr;
  int ndim = 0;
  int shape[4];

  void toCPU();
  void toGPU();
};

Tensor::Tensor(std::vector<int> shape_) {
  // reshape
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = shape_[i]; }
  int N_ = num_elem();
  //reshape fin

  // buf = (float *) calloc(N_, sizeof(float));
  CHECK_CUDA(hipHostMalloc(&buf, N_ * sizeof(float)));
  CHECK_CUDA(hipMalloc(&gbuf, N_ * sizeof(float)));
}

Tensor::Tensor(std::vector<int> shape_, float *buf_) {
  // reshape
  ndim = shape_.size();
  for (int i = 0; i < ndim; ++i) { shape[i] = shape_[i]; }
  int N_ = num_elem();
  // reshape fin

  // buf = (float *) calloc(N_, sizeof(float));
  CHECK_CUDA(hipHostMalloc(&buf, N_ * sizeof(float)));
  CHECK_CUDA(hipMalloc(&gbuf, N_ * sizeof(float)));
  memcpy(buf, buf_, N_ * sizeof(float)); // for (int n = 0; n < N_; ++n) { buf[n] = buf_[n]; }
  CHECK_CUDA(hipMemcpy(gbuf, buf_, N_ * sizeof(float), hipMemcpyHostToDevice));
}

void Tensor::toCPU(){
  CHECK_CUDA(hipMemcpy(buf, gbuf, num_elem() * sizeof(float), hipMemcpyDeviceToHost));
}

void Tensor::toGPU(){
  CHECK_CUDA(hipMemcpy(gbuf, buf, num_elem() * sizeof(float), hipMemcpyHostToDevice));
}

Tensor::~Tensor() {
  // if (buf != nullptr) free(buf);
  hipHostFree(buf);
  CHECK_CUDA(hipFree(gbuf));
}

int Tensor::num_elem() {
  int sz = 1;
  for (int i = 0; i < ndim; ++i) { sz *= shape[i]; }
  return sz;
}

void Tensor::fill_zeros() {
  int N_ = num_elem();
  for (int n = 0; n < N_; ++n) { buf[n] = 0.0; }
}

// Parameters
Tensor *w_conv1, *w_conv2, *w_conv3, *w_conv4, *w_conv5, *w_conv6, *b_conv1,
    *b_conv2, *b_conv3, *b_conv4, *b_conv5, *b_conv6, *w_fc1, *w_fc2, *w_fc3,
    *b_fc1, *b_fc2, *b_fc3, *gamma_conv1, *beta_conv1, *gamma_conv6, *beta_conv6;

// Activations
Tensor *a_conv1, *a_layernorm1, *a_relu1, *a_pool1;
Tensor *a_conv2, *a_relu2, *a_pool2;
Tensor *a_conv3, *a_relu3;
Tensor *a_conv4, *a_relu4;
Tensor *a_conv5, *a_relu5;
Tensor *a_conv6, *a_layernorm6, *a_relu6, *a_pool6;
Tensor *a_collapse;
Tensor *a_linear1, *a_relu7;
Tensor *a_linear2, *a_relu8;
Tensor *a_linear3;
//me
Tensor *a_output;

// Operations
void conv1d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride, int padding, int dilation, bool has_bias);
void relu(Tensor *input, Tensor *output);
void maxpool1d(Tensor *input, Tensor *output, int kernel_size, int stride);
void collapse(Tensor *input, Tensor *output);
void linear(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            bool has_bias);
void layernorm(Tensor *input, Tensor *gamma, Tensor *beta, Tensor *output);
//me
void find_maxIdx(Tensor *input, Tensor *output, int n);

void check(Tensor *t_b, Tensor *t_bb){
  t_b->toCPU();
  t_bb->toCPU();
  int count = 0;
  printf("no batch : %d, batch : %d\n", t_b->num_elem(), t_bb->num_elem());
  for (int i=0; i<t_b->num_elem(); ++i){
    if (t_b->buf[i] != t_bb->buf[i]){
      printf("%d : %f <-> %f\n", i, t_b->buf[i], t_bb->buf[i]);
      count++;
      if(count >= 10) break;
    }
  }
  printf("\n finish \n");
}

void why0(Tensor *t){
  t->toCPU();
  bool ok = true;
  int count = 0;
  for(int i = 0; i < t->num_elem(); i++){
    printf(" %f", t->buf[i]);
    if(t->buf[i] == 0.0f){
      count++;
    }else{
      count = 0;
    }
    if(count == 100){
      ok = false;
      printf("here!!!!!!!! %d\n", i);
      break;
    }
  }
  if(ok) printf("\n*****successful %d******\n", t->num_elem());
}

// Only the first process (root, mpi_rank == 0) has the input and output
// Parallelization method is totally up to you, but you should gather 
// the output at rank 0
void classifier(float *input_, float *output_, int N) {
  // if (mpi_rank == 0) {
    int loop = (BATCH + N - 1) / BATCH;
    for (int idx = 0; idx < loop; ++idx) {  // N input sentences

      // Load one input sentence from input
      Tensor *one_input = new Tensor({BATCH, 1, VOCAB_SIZE, MAX_LENGTH}, input_ + idx * BATCH * VOCAB_SIZE * MAX_LENGTH);

      // yelim!!
      CHECK_CUDA(hipMemcpy(one_input->gbuf, input_ + idx * BATCH * VOCAB_SIZE * MAX_LENGTH, BATCH * VOCAB_SIZE * MAX_LENGTH * sizeof(float), hipMemcpyHostToDevice));
      
      // Conv block 1 : Conv1d + LayerNorm + ReLU + MaxPool1d
      conv1d(one_input, w_conv1, b_conv1, a_conv1, 1, 0, 1, true);
      layernorm(a_conv1, gamma_conv1, beta_conv1, a_layernorm1);
      relu(a_layernorm1, a_relu1);
      maxpool1d(a_relu1, a_pool1, 3, 3);

      // Conv block 2 : Conv1d + ReLU + MaxPool1d
      conv1d(a_pool1, w_conv2, b_conv2, a_conv2, 1, 0, 1, true);
      relu(a_conv2, a_relu2);
      maxpool1d(a_relu2, a_pool2, 3, 3);
      
      // Conv block 3 : Conv1d + ReLU
      conv1d(a_pool2, w_conv3, b_conv3, a_conv3, 1, 0, 1, true);
      relu(a_conv3, a_relu3);
      
      // Conv block 4 : Conv1d + ReLU
      conv1d(a_relu3, w_conv4, b_conv4, a_conv4, 1, 0, 1, true);
      relu(a_conv4, a_relu4);

      // Conv block 5 : Conv1d + ReLU
      conv1d(a_relu4, w_conv5, b_conv5, a_conv5, 1, 0, 1, true);
      relu(a_conv5, a_relu5);

      // Conv block 6 : Conv1d + LayerNorm + ReLU + MaxPool1d
      conv1d(a_relu5, w_conv6, b_conv6, a_conv6, 1, 0, 1, true);
      layernorm(a_conv6, gamma_conv6, beta_conv6, a_layernorm6);
      relu(a_layernorm6, a_relu6);
      maxpool1d(a_relu6, a_pool6, 3, 3);
      
      // Collapse
      collapse(a_pool6, a_collapse);
      
      // FC block 1 : Linear + ReLU
      linear(a_collapse, w_fc1, b_fc1, a_linear1, true);
      relu(a_linear1, a_relu7);

      // FC block 2 : Linear + ReLU
      linear(a_relu7, w_fc2, b_fc2, a_linear2, true);
      relu(a_linear2, a_relu8);

      // FC block 3 : Linear
      linear(a_relu8, w_fc3, b_fc3, a_linear3, true);
      a_linear3->toCPU();
      // find_maxIdx(a_linear3, a_output, idx);
      for(int b = 0; b < BATCH; b++){
        float max_val = -1e99f;
        int max_idx = 0;
        int num = a_linear3->num_elem() / BATCH;
        for (int i = 0; i < num; ++i) {
          if (a_linear3->buf[b * num + i] > max_val) {
            max_val = a_linear3->buf[b * num + i];
            max_idx = i;
          }
        }
        if(idx * BATCH + b >= N) break;
        output_[idx * BATCH + b] = max_idx;
      }
      // CHECK_CUDA(hipMemcpy(output_ + BATCH * idx, a_output->gbuf, BATCH * sizeof(float), hipMemcpyDeviceToHost));
      // CHECK_CUDA(hipDeviceSynchronize());
    }
    //}  // end N input sentences loop
  // }    // if mpi_rank == 0
}

__global__ void conv1d_kernel(float *in, float *out, float *weight, float *bias, int out_channels, int in_channels, int kernel_size, int input_length, int output_length, bool has_bias){
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int b = tidx / (out_channels * output_length);
  int oc = (tidx / output_length) % out_channels;
  int ol = tidx % output_length;

  if(oc >= out_channels || ol >= output_length) return;  
  
  float val = 0.0f;
  int offset = ol;
  for (int ic = 0; ic < in_channels; ++ic) {
    for (int ks = 0; ks < kernel_size; ++ks) {
      val += weight[oc * in_channels * kernel_size + ic * kernel_size + ks] *
                 in[b * in_channels * input_length + ic * input_length + ks + offset];
    }
  }
  if (has_bias) val += bias[oc];
  out[b * out_channels * output_length + oc * output_length + ol] = val;  
}

void conv1d(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            int stride = 1, int padding = 0, int dilation = 1,
            bool has_bias = true) {
  float *in = input->gbuf;
  float *out = output->gbuf;
  float *w = weight->gbuf;
  float *b = bias->gbuf;

  int out_channels = weight->shape[0];
  int in_channels = weight->shape[1];
  int kernel_size = weight->shape[2];
  int input_length = input->shape[3];
  int output_length =
      (input->shape[3] + 2 * padding - dilation * (kernel_size - 1) - 1) / stride + 1;

  int total_threads = BATCH * out_channels * output_length;
  int block_size = 1024; 
  dim3 blockDim(block_size);
  dim3 gridDim((total_threads + block_size - 1) / block_size);
  conv1d_kernel<<<gridDim, blockDim>>>(in, out, w, b, out_channels, in_channels, kernel_size, input_length, output_length, has_bias);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void relu_kernel(float *in, float *out, int N){
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  if(tidx >= N) return;
  out[tidx] = fmaxf(in[tidx], 0.0f);
}

void relu(Tensor *input, Tensor *output) {
  float *in = input->gbuf;
  float *out = output->gbuf;
  int N = input->num_elem();

  relu_kernel<<<CEIL_DIV(N, 256), 256>>>(in, out, N);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void maxpool1d_kernel(float *in, float *out, int IL, int OC, int OL, int kernel_size, int stride){
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int b = tidx / (OL * OC);
  int oc = (tidx / OL) % OC;
  int ol = tidx % OL;
  if(oc >= OC || ol >= OL) return;

  int i_idx = b * OC * IL + oc * IL + ol * stride;
  int o_idx = b * OC * OL + oc * OL + ol;
  float mx = -1e99;
  for (int ks = 0; ks < kernel_size; ++ks) {
    float val = in[i_idx + ks];
    if (val > mx) mx = val;
  }
  out[o_idx] = mx;
}

void maxpool1d(Tensor *input, Tensor *output, int kernel_size, int stride) {
  float *in = input->gbuf;
  float *out = output->gbuf;

  int IL = input->shape[3];
  int OC = output->shape[2];
  int OL = output->shape[3];

  int total_threads = BATCH * OC * OL;
  int block_size = 512;
  dim3 blockDim(block_size);
  dim3 gridDim((total_threads + block_size - 1) / block_size);
  maxpool1d_kernel<<<gridDim, blockDim>>>(in, out, IL, OC, OL, kernel_size, stride);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void collapse_kernel(float *in, float *out, int N){
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if(n >= N) return;
  out[n] = in[n];
}

void collapse(Tensor *input, Tensor *output) {
  float *in = input->gbuf;
  float *out = output->gbuf;
  int N = input->num_elem();

  collapse_kernel<<<CEIL_DIV(N, 256), 256>>>(in, out, N);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void linear_kernel(float *in, float *out, float *weight, float *bias, int IC, int OC, bool has_bias){
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  int b = tidx / OC;
  int oc = tidx % OC;

  if(oc >= OC) return;

  float val = 0.0;
  for (int ic = 0; ic < IC; ++ic) {
    val += in[b * IC + ic] * weight[oc * IC + ic];
  }
  if (has_bias) val += bias[oc];
  out[b * OC + oc] = val;
}

void linear(Tensor *input, Tensor *weight, Tensor *bias, Tensor *output,
            bool has_bias) {
  float *in = input->gbuf;
  float *out = output->gbuf;
  float *w = weight->gbuf;
  float *b = bias->gbuf;

  int IC = input->shape[2];
  int OC = output->shape[2];

  int total_threads = BATCH * OC;
  int block_size = 512;
  dim3 blockDim(block_size);
  dim3 gridDim((total_threads + block_size - 1) / block_size);
  linear_kernel<<<gridDim, blockDim>>>(in, out, w, b, IC, OC, has_bias);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void layernorm_kernel(float *in, float *out, float *gamma, float *bias, int N){
  int b = blockDim.y * blockIdx.y + threadIdx.y;
  int idx = b * N;

  // E[X], E[X^2]
  float sum1 = 0.0f, sum2 = 0.0f;
  for (int i = 0; i < N; ++i) {
      sum1 += in[idx + i];
      sum2 += in[idx + i] * in[idx + i];
  }
  float mean1 = sum1 / (float)N;
  float mean2 = sum2 / (float)N;

  // V[X]
  float var = mean2 - mean1 * mean1;  

  // Normalization
  for (int i = 0; i < N; ++i) {
    out[idx + i] = (in[idx + i] - mean1) / sqrtf(var + 1e-5) * gamma[i] + bias[i];
  }
}

void layernorm(Tensor *input, Tensor *gamma, Tensor *beta, Tensor *output) {
  float *in = input->gbuf;
  float *out = output->gbuf;
  float *g = gamma->gbuf;
  float *b = beta->gbuf;
  int N = input->num_elem() / BATCH;

  dim3 block(1, 1);
  dim3 grid(1, BATCH);
  layernorm_kernel<<<grid, block>>>(in, out, g, b, N);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
}

__global__ void find_maxIdx_kernel(float *in, float *out, int N, int idx){
  for(int b = 0; b < BATCH; b++){
    float max_val = -1e99f;
    int max_idx = 0;
    for (int i = 0; i < N; ++i) {
      if (in[b * N + i] > max_val) {
        max_val = in[b * N + i];
        max_idx = i;
      }
    }
    out[idx * BATCH + b] = max_idx;
    if(idx * BATCH + b >= N) break;
  }
}

void find_maxIdx(Tensor *input, Tensor *output, int idx) {
  float *in = input->gbuf;
  float *out = output->gbuf;
  int N = input->num_elem() / BATCH;

  find_maxIdx_kernel<<<1, 1>>>(in, out, N, idx);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());
}

// load the parameter binary file and store parameters into Tensors
// Only the first process (root, mpi_rank == 0) has the parameter
// You must broadcast it to the others
void initialize_classifier(float *parameter, int N) {
  // MPI_Comm_rank(MPI_COMM_WORLD, &mpi_rank);
  // if (mpi_rank == 0) {
    w_conv1 = new Tensor({256, 70, 7}, parameter + OFFSET0);
    b_conv1 = new Tensor({256}, parameter + OFFSET1);
    gamma_conv1 = new Tensor({256, 1008}, parameter + OFFSET2);
    beta_conv1 = new Tensor({256, 1008}, parameter + OFFSET3);
    w_conv2 = new Tensor({256, 256, 7}, parameter + OFFSET4);
    b_conv2 = new Tensor({256}, parameter + OFFSET5);
    w_conv3 = new Tensor({256, 256, 3}, parameter + OFFSET6);
    b_conv3 = new Tensor({256}, parameter + OFFSET7);
    w_conv4 = new Tensor({256, 256, 3}, parameter + OFFSET8);
    b_conv4 = new Tensor({256}, parameter + OFFSET9);
    w_conv5 = new Tensor({256, 256, 3}, parameter + OFFSET10);
    b_conv5 = new Tensor({256}, parameter + OFFSET11);
    w_conv6 = new Tensor({256, 256, 3}, parameter + OFFSET12);
    b_conv6 = new Tensor({256}, parameter + OFFSET13);
    gamma_conv6 = new Tensor({256, 102}, parameter + OFFSET14);
    beta_conv6 = new Tensor({256, 102}, parameter + OFFSET15);
    w_fc1 = new Tensor({1024, 8704}, parameter + OFFSET16);
    b_fc1 = new Tensor({1024}, parameter + OFFSET17);
    w_fc2 = new Tensor({1024, 1024}, parameter + OFFSET18);
    b_fc2 = new Tensor({1024}, parameter + OFFSET19);
    w_fc3 = new Tensor({4, 1024}, parameter + OFFSET20);
    b_fc3 = new Tensor({4}, parameter + OFFSET21);

    a_conv1 = new Tensor({BATCH, 1, 256, 1008});
    a_layernorm1 = new Tensor({BATCH, 1, 256, 1008});
    a_relu1 = new Tensor({BATCH, 1, 256, 1008});
    a_pool1 = new Tensor({BATCH, 1, 256, 336});
    a_conv2 = new Tensor({BATCH, 1, 256, 330});
    a_relu2 = new Tensor({BATCH, 1, 256, 330});
    a_pool2 = new Tensor({BATCH, 1, 256, 110});
    a_conv3 = new Tensor({BATCH, 1, 256, 108});
    a_relu3 = new Tensor({BATCH, 1, 256, 108});
    a_conv4 = new Tensor({BATCH, 1, 256, 106});
    a_relu4 = new Tensor({BATCH, 1, 256, 106});
    a_conv5 = new Tensor({BATCH, 1, 256, 104});
    a_relu5 = new Tensor({BATCH, 1, 256, 104});
    a_conv6 = new Tensor({BATCH, 1, 256, 102});
    a_layernorm6 = new Tensor({BATCH, 1, 256, 102});
    a_relu6 = new Tensor({BATCH, 1, 256, 102});
    a_pool6 = new Tensor({BATCH, 1, 256, 34});
    a_collapse = new Tensor({BATCH, 1, 8704});
    a_linear1 = new Tensor({BATCH, 1, 1024});
    a_relu7 = new Tensor({BATCH, 1, 1024});
    a_linear2 = new Tensor({BATCH, 1, 1024});
    a_relu8 = new Tensor({BATCH, 1, 1024});
    a_linear3 = new Tensor({BATCH, 1, 4});

    //yelim
    a_output = new Tensor({BATCH, 1});
  // }
}

// Free all dynamically allocated variables
void finalize_classifier() {
  // if (mpi_rank == 0) {
    delete w_conv1;
    delete b_conv1;
    delete w_conv2;
    delete b_conv2;
    delete w_conv3;
    delete b_conv3;
    delete w_conv4;
    delete b_conv4;
    delete w_conv5;
    delete b_conv5;
    delete w_conv6;
    delete b_conv6;
    delete w_fc1;
    delete b_fc1;
    delete w_fc2;
    delete b_fc2;
    delete w_fc3;
    delete b_fc3;
    delete gamma_conv1;
    delete gamma_conv6;
    delete beta_conv1;
    delete beta_conv6;
    delete a_conv1;
    delete a_layernorm1;
    delete a_relu1;
    delete a_pool1;
    delete a_conv2;
    delete a_relu2;
    delete a_pool2;
    delete a_conv3;
    delete a_relu3;
    delete a_conv4;
    delete a_relu4;
    delete a_conv5;
    delete a_relu5;
    delete a_conv6;
    delete a_layernorm6;
    delete a_relu6;
    delete a_pool6;
    delete a_collapse;
    delete a_linear1;
    delete a_relu7;
    delete a_linear2;
    delete a_relu8;
    delete a_linear3;
    //yelim
    delete a_output;
  // }
}